#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



#define THREADS_PER_BLOCK 256


__global__ void add(int *in1, int *in2, int *out){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  out[index] = in1[index] + in2[index];
}

void random_ints(int *i, int size){
  for(int k=0; k<size; k++){
    i[k] = rand()%50;
  }
}

// int* testmain(int num){
int* vectoradd(int num){

  int size = num * sizeof(int);

  int *in1, *in2, *out; // host copies
  in1 = (int *)malloc(size);
  in2 = (int *)malloc(size);
  out = (int *)malloc(size);

  int *d_in1, *d_in2, *d_out; // device copies
  hipMalloc((void **)&d_in1, size);
  hipMalloc((void **)&d_in2, size);
  hipMalloc((void **)&d_out, size);

  // generate random numbers
  random_ints(in1, num);
  random_ints(in2, num);
  
  // copy host data to device
  hipMemcpy(d_in1, in1, size, hipMemcpyHostToDevice);
  hipMemcpy(d_in2, in2, size, hipMemcpyHostToDevice);
  
  // run kernel
  add<<<num/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_in1, d_in2, d_out);
  hipDeviceSynchronize();

  // copy device data to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

  // cleanup
  free(in1);
  free(in2);
  free(out);
  hipFree(d_in1);
  hipFree(d_in2);
  hipFree(d_out);

  return out;
}
