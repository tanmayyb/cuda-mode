#include "hip/hip_runtime.h"
#include <dot.hpp>
#include <iostream>
#include <vector>
// #include <Eigen/Core>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256

inline void HandleError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at " << file << ":" << line << " code=" << err
              << " \"" << hipGetErrorString(err) << "\"" << std::endl;
    exit(EXIT_FAILURE);
  }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

// __global__ void CUDAvecadd(
//   Eigen::Vector3f *v1,
//   Eigen::Vector3f *v2,
//   float *out,
//   size_t N
// ){
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   if(index >= N) return;
//   out[index] = v1[index].dot(v2[index]);
// }


// std::vector<float> launch_vecadd(
//   std::vector<Eigen::Vector3f> &v1, 
//   std::vector<Eigen::Vector3f> &v2
// ){

//   int size = v1.size();
//   float *ret = new float[size];

//   Eigen::Vector3f *d_v1, *d_v2; // device copy
//   float *d_ret;

//   HANDLE_ERROR(hipMalloc((void **)&d_v1, sizeof(Eigen::Vector3f)*size));
//   HANDLE_ERROR(hipMalloc((void **)&d_v2, sizeof(Eigen::Vector3f)*size));
//   HANDLE_ERROR(hipMalloc((void **)&d_ret, sizeof(float)*size));

//   HANDLE_ERROR(hipMemcpy(d_v1, v1.data(), sizeof(Eigen::Vector3f)*size, hipMemcpyHostToDevice));
//   HANDLE_ERROR(hipMemcpy(d_v2, v2.data(), sizeof(Eigen::Vector3f)*size, hipMemcpyHostToDevice));

//   CUDAvecadd<<<size/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_v1, d_v2, d_ret, size);
//   hipDeviceSynchronize();
  
//   HANDLE_ERROR(hipMemcpy(ret, d_ret, sizeof(float)*size, hipMemcpyDeviceToHost));
//   std::vector<float> vec(ret, ret + size);

//   free(ret);
//   HANDLE_ERROR(hipFree(d_v1));
//   HANDLE_ERROR(hipFree(d_v2));
//   HANDLE_ERROR(hipFree(d_ret));
//   return vec;
// }


void hello_world(){
  std::cout<<"hello world!"<<std::endl;
}