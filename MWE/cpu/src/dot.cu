#include "hip/hip_runtime.h"
#include <dot.hpp>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
// #include <thrust/device_vector.h>
// #include <thrust/transform_reduce.h>
// #include <thrust/functional.h>

#define THREADS_PER_BLOCK 256
// #define THREADS_PER_BLOCK 50



// dot produc
__global__ void computeDotProduct(
  Vector3f* d_vec1, 
  Vector3f* d_vec2, 
  float* d_dotvec, 
  float* d_result, 
  int N
){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index >= N) return;
  float result = d_vec1[index].getX() * d_vec2[index].getX()
                  + d_vec1[index].getY() * d_vec2[index].getY()
                  + d_vec1[index].getZ() * d_vec2[index].getZ();
   
  d_dotvec[index] =  result;
  atomicAdd(d_result, result);
}


float launch_vecdot(
  std::vector<Vector3f> *h_vec1, 
  std::vector<Vector3f> *h_vec2
){

  Vector3f *d_vec1, *d_vec2;
  float *d_dotvec, *d_result;

  int size = (*h_vec1).size() * sizeof(Vector3f);
  hipMalloc((void**)&d_vec1, size);
  hipMalloc((void**)&d_vec2, size);
  hipMalloc((void**)&d_dotvec, (*h_vec1).size());
  hipMalloc((void**)&d_result, 1*sizeof(float));
  hipMemcpy(d_vec1, (*h_vec1).data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_vec2, (*h_vec2).data(), size, hipMemcpyHostToDevice);

  // run kernel
  computeDotProduct<<<size/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_vec1, d_vec2, d_dotvec, d_result, size);
  // computeDotProduct<<<1, THREADS_PER_BLOCK>>>(d_vec1, d_vec2, d_dotvec, d_result, size);

  hipDeviceSynchronize();

  float h_result;
  hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

  // cleanup
  hipFree(d_vec1);
  hipFree(d_vec2);
  hipFree(d_dotvec);
  hipFree(d_result);

  return h_result;
}


void hello_world(){
  std::cout<<"hello world!"<<std::endl;
}