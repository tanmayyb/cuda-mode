#include <dot.hpp>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>

#define THREADS_PER_BLOCK 256

struct dot_product_functor {
  __host__ __device__ float operator()(const Vector3& obj) const {
    return  obj.getX() * obj.getX() +
            obj.getY() * obj.getY() +
            obj.getZ() * obj.getZ();
  }
};



float launch_vecdot(
  std::vector<Vector3> *h_v1, 
  std::vector<Vector3> *h_v2
){

  // int size = (*h_v1).size();
  // float *h_ret = new float[size];
  // float *d_v1, *d_v2;
  // float *d_ret;

  thrust::device_vector<Vector3> d_objects = *h_v1;
  
  float total_dot_product = thrust::transform_reduce(
      d_objects.begin(), d_objects.end(),
      dot_product_functor(), // Functor to compute dot product for each object
      0.0f,                  // Initial value for the reduction
      thrust::plus<float>()   // Binary operation to sum results
  );

  // std::vector<float> ret;
  return total_dot_product;
}


void hello_world(){
  std::cout<<"hello world!"<<std::endl;
}