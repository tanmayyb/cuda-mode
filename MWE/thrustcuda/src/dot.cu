#include <dot.hpp>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>

#define THREADS_PER_BLOCK 256

// struct Vector3fDotProduct {
//   __host__ __device__ float operator()(
//     const Vector3f& a, 
//     const Vector3f& b
//   ) const {
//     return  a.getX() * b.getX() + 
//             a.getY() * b.getY() + 
//             a.getZ() * b.getZ();
//   }
// };

struct Vector3fDotProduct {
  __host__ __device__ float operator()(
    // const Vector3f& a, 
    // const Vector3f& b
    const thrust::tuple<Vector3f, Vector3f>& t
  ) const {

    auto a = thrust::get<0>(t);
    auto b = thrust::get<1>(t);
    return  a.getX() * b.getX() + 
            a.getY() * b.getY() + 
            a.getZ() * b.getZ();
  }
};



float computeDotProductThrust(
  const std::vector<Vector3f>& h_vec1, 
  const std::vector<Vector3f>& h_vec2
) {
  int n = h_vec1.size();

  // Create thrust::device_vector from std::vector
  thrust::device_vector<Vector3f> d_vec1(h_vec1.begin(), h_vec1.end());
  thrust::device_vector<Vector3f> d_vec2(h_vec2.begin(), h_vec2.end());

  // // Use thrust::transform_reduce to compute dot product
  // float result = thrust::transform_reduce(
  //   d_vec1.begin(), 
  //   d_vec1.end(), 
  //   // d_vec2.begin(),               
  //   Vector3fDotProduct(),         
  //   0.0f,                         
  //   thrust::plus<float>()         
  // );

  auto first = thrust::make_zip_iterator(thrust::make_tuple(d_vec1.begin(), d_vec2.begin()));
  auto last = thrust::make_zip_iterator(thrust::make_tuple(d_vec1.end(), d_vec2.end()));

  float result = thrust::transform_reduce(
    first,
    last,
    Vector3fDotProduct(),
    0.0f,
    thrust::plus<float>()
  );

  return result;
}


float launch_vecdot(
  std::vector<Vector3f> *h_vec1, 
  std::vector<Vector3f> *h_vec2
){

  // Calculate dot product using Thrust
  float total_dot_product = computeDotProductThrust(*h_vec1, *h_vec2);

  std::cout << "Dot Product (Thrust): " << total_dot_product << std::endl;
  return total_dot_product;
}


void hello_world(){
  std::cout<<"hello world!"<<std::endl;
}