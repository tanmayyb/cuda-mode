#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <chrono>

#include <algorithm>
#include <ctime>


__global__ void CUDAvecadd(
    int * a,
    int * b,
    int * result,
    int t
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<t){
        result[i] = a[i] + b[i];
    }
}


std::vector<std::vector<int>> generate_vectors(){
    // ready the vectors
    int len_vec = 134217728; // 2^27
    int seed = 0;
    std::srand(unsigned(seed));
    std::vector<int> u(len_vec);
    std::generate(u.begin(), u.end(), std::rand);
    std::vector<int> v = u;
    return std::vector<std::vector<int>> {u,v};
}


int main(){
    auto vectors = generate_vectors();
    int* u = vectors[0].data();
    int* v = vectors[1].data();
    int t = vectors[0].size();
    size_t t_ = t * sizeof(int);

    int nthreads = 256;
    int nblocks = t/nthreads;
    // // perform and measure performance
    auto start = std::chrono::high_resolution_clock::now();
    // std::vector<int> result(t);
    int* result;
    hipMalloc(&result, t_);
    CUDAvecadd<<<nblocks, nthreads>>>(u, v, result, t);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "Function execution time: " << duration.count() << " seconds" << std::endl;

    return 0;
}